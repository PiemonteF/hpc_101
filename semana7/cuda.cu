#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <time.h>

// Kernel to compute partial sums on the GPU
__global__ void partial_sum(int *d_vetor, long long *d_partial_sums, int N) {
    extern __shared__ long long sdata[];
    int tid = threadIdx.x; // Thread ID within the block
    int i = blockIdx.x * blockDim.x + threadIdx.x; // Global thread ID

    // Load elements into shared memory
    sdata[tid] = (i < N) ? d_vetor[i] : 0;
    __syncthreads();

    // Perform reduction in shared memory
    for (int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (tid < s) {
            sdata[tid] += sdata[tid + s];
        }
        __syncthreads();
    }

    // Write the result of this block to the partial sums array
    if (tid == 0) {
        d_partial_sums[blockIdx.x] = sdata[0];
    }
}

int main() {
    // Vector sizes for testing
    int tamanhos[4] = {1000000, 10000000, 50000000, 100000000};
    int num_testes = 4;

    for (int t = 0; t < num_testes; t++) {
        int N = tamanhos[t];
        int *vetor = (int*) malloc(N * sizeof(int));

        // Initialize random number generator
        srand(time(NULL));
        for (int i = 0; i < N; i++) {
            vetor[i] = rand() % 100; // Random numbers between 0 and 99
        }

        // Device memory allocation
        int *d_vetor;
        long long *d_partial_sums;
        int block_size = 256;
        int num_blocks = (N + block_size - 1) / block_size;
        hipMalloc((void**)&d_vetor, N * sizeof(int));
        hipMalloc((void**)&d_partial_sums, num_blocks * sizeof(long long));

        // Copy data to device
        hipMemcpy(d_vetor, vetor, N * sizeof(int), hipMemcpyHostToDevice);

        // Start timer
        hipEvent_t start, stop;
        hipEventCreate(&start);
        hipEventCreate(&stop);
        hipEventRecord(start);

        // Launch kernel
        partial_sum<<<num_blocks, block_size, block_size * sizeof(long long)>>>(d_vetor, d_partial_sums, N);

        // Copy partial sums back to host
        long long *partial_sums = (long long*) malloc(num_blocks * sizeof(long long));
        hipMemcpy(partial_sums, d_partial_sums, num_blocks * sizeof(long long), hipMemcpyDeviceToHost);

        // Compute final sum on CPU
        long long soma = 0;
        for (int i = 0; i < num_blocks; i++) {
            soma += partial_sums[i];
        }

        // Stop timer
        hipEventRecord(stop);
        hipEventSynchronize(stop);
        float milliseconds = 0;
        hipEventElapsedTime(&milliseconds, start, stop);

        // Output results
        printf("Teste %d - Tamanho do vetor: %d\n", t+1, N);
        printf("Soma Paralela com GPU: %lld\n", soma);
        printf("Tempo de execução com CUDA: %f segundos\n\n", milliseconds / 1000);

        // Free memory
        free(vetor);
        free(partial_sums);
        hipFree(d_vetor);
        hipFree(d_partial_sums);
    }

    return 0;
}
